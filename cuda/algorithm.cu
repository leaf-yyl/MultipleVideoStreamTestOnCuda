#include "hip/hip_runtime.h"


#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"

#include "algorithm.cuh"
#include "utils/scimage.h"

__device__ u_char clamp(float t)
{
    if (t < 0) {
        return 0;
    } else if (t > 255){
        return 255;
    }

    return t;
}

inline unsigned int divUpper(int l, int dimension)
{
    return (l - 1) / dimension + 1;
}

__global__ void
kernel_colorSpaceYUV420PToRGBA(dev_t *src, dev_t *dst, int pitch_src, int pitch_dst, int w, int h)
{
    unsigned int dim_x = blockDim.x * blockIdx.x + threadIdx.x;
    unsigned int dim_y = blockDim.y * blockIdx.y + threadIdx.y;

    int r,g,b,y,u,v;
    if (dim_x < w && dim_y < h) {
        y = *((u_char*)src + dim_y * pitch_src + dim_x);
        u = *((u_char*)src + (h + dim_y / 4) * pitch_src + dim_x / 2);
        v = *((u_char*)src + (h * 5 + dim_y) / 4 * pitch_src + dim_x / 2);
        r = clamp(y + 1.402 * (v - 128) + 0.5);
        g = clamp(y - 0.34414 * (u - 128) - 0.71414 * (v - 128) + 0.5);
        b = clamp(y + 1.772 * (u - 128) + 0.5);
//        *((uint32_t*)dst + dim_y * pitch_dst / 4 + dim_x) = (r << 24) + (g << 16) + (b << 8);
        *((u_char*)dst + dim_y * pitch_dst + dim_x * 4) = r;
        *((u_char*)dst + dim_y * pitch_dst + dim_x * 4 + 1) = g;
        *((u_char*)dst + dim_y * pitch_dst + dim_x * 4 + 2) = b;
        *((u_char*)dst + dim_y * pitch_dst + dim_x * 4 + 3) = 255;
    }
}

__global__ void
kernel_colorSpaceRGBAToYUV420P(dev_t *src, dev_t *dst, int pitch_src, int pitch_dst, int w, int h)
{
    unsigned int dim_x = blockDim.x * blockIdx.x + threadIdx.x;
    unsigned int dim_y = blockDim.y * blockIdx.y + threadIdx.y;

    int r,g,b;
    unsigned int rgba;
    if (dim_x < w && dim_y < h) {
        rgba = *((uint32_t*)dst + dim_y * pitch_dst / 4 + dim_x);
        r = (rgba >> 24);
        g = (rgba >> 16) & 0xff;
        b = (rgba >> 8) & 0xff;
        *((u_char*)src + dim_y * pitch_src + dim_x) = clamp(0.299 * r + 0.587 * g + 0.114 * b);
        *((u_char*)src + (h + dim_y / 4) * pitch_src + dim_x / 2) = clamp(-0.1687 * r - 0.3313 * g + 0.5 * b + 128);
        *((u_char*)src + (h * 5 + dim_y) / 4 * pitch_src + dim_x / 2) = clamp(0.5 * r - 0.4187 * g - 0.0813 * b + 128);
    }
}

void ColorSpaceConvertion(ScGPUImage *src, ScGPUImage *dst)
{
    dim3 block(8, 8);
    dim3 grid((src->m_pixel_width - 1) / block.x + 1, (src->m_pixel_height - 1) / block.y + 1);
    if (ScGPUImage::ScPixFormat_YUV420P == src->m_pix_fmt)
    {
        switch (dst->m_pix_fmt)
        {
        case ScGPUImage::ScPixFormat_YUV420P:
            hipMemcpy2D(dst->m_dev_addr, dst->m_data_width, src->m_dev_addr, src->m_data_width,
                         src->m_pixel_width, src->m_pixel_height * 3 / 2, hipMemcpyDeviceToDevice);
            break;
        case ScGPUImage::ScPixFormat_RGBA:
            kernel_colorSpaceYUV420PToRGBA<<<grid, block>>>(src->m_dev_addr, dst->m_dev_addr,
                                                               src->m_data_width, dst->m_data_width,
                                                               src->m_pixel_width, src->m_pixel_height);
            break;
        }
    } else if (ScGPUImage::ScPixFormat_RGBA == src->m_pix_fmt){
        switch (dst->m_pix_fmt)
        {
        case ScGPUImage::ScPixFormat_YUV420P:
            kernel_colorSpaceRGBAToYUV420P<<<grid, block, 0>>>(src->m_dev_addr, dst->m_dev_addr,
                                                               src->m_data_width, dst->m_data_width,
                                                               src->m_pixel_width, src->m_pixel_height);
            break;
        case ScGPUImage::ScPixFormat_RGBA:
            hipMemcpy2D(dst->m_dev_addr, dst->m_data_width, src->m_dev_addr, src->m_data_width,
                         src->m_pixel_width * 4, src->m_pixel_height, hipMemcpyDeviceToDevice);
            break;
        }
    }
}

__global__ void
kernel_horizontalReversal(dev_t *src, dev_t *dst, uint pitch_src, uint pitch_dst, uint pixel_w, uint pixel_h)
{
    unsigned int dim_x = blockDim.x * blockIdx.x + threadIdx.x;
    unsigned int dim_y = blockDim.y * blockIdx.y + threadIdx.y;

    if (dim_x < pixel_w && dim_y < pixel_h) {
        unsigned int rgba = *((uint32_t*)src + dim_y * pitch_src / 4 + dim_x);
        *((uint32_t*)dst + dim_y * pitch_dst / 4 + pixel_w - dim_x) = rgba;
    }
}

__global__ void
kernel_verticalReversal(dev_t *src, dev_t *dst, uint pitch_src, uint pitch_dst, uint pixel_w, uint pixel_h)
{
    unsigned int dim_x = blockDim.x * blockIdx.x + threadIdx.x;
    unsigned int dim_y = blockDim.y * blockIdx.y + threadIdx.y;

    if (dim_x < pixel_w && dim_y < pixel_h) {
        unsigned int rgba = *((uint32_t*)src + dim_y * pitch_src / 4 + dim_x);
        *((uint32_t*)dst + (pixel_h - dim_y) * pitch_dst / 4 + dim_x) = rgba;
    }
}

void HorizontalReversalRGBA(ScGPUImage *src, ScGPUImage *dst)
{
    dim3 block(8, 8);
    dim3 grid(divUpper(src->m_pixel_width, block.x), divUpper(src->m_pixel_height, block.y));
    kernel_horizontalReversal<<<grid, block>>>(src->m_dev_addr, dst->m_dev_addr, src->m_data_width, dst->m_data_width,
                                               src->m_pixel_width, src->m_pixel_height);

}

void VerticalReversalRGBA(ScGPUImage *src, ScGPUImage *dst)
{
    dim3 block(8, 8);
    dim3 grid(divUpper(src->m_pixel_width, block.x), divUpper(src->m_pixel_height, block.y));
    kernel_verticalReversal<<<grid, block>>>(src->m_dev_addr, dst->m_dev_addr, src->m_data_width, dst->m_data_width,
                                             src->m_pixel_width, src->m_pixel_height);
}

__global__ void
kernel_renderRGBA2Surface(hipSurfaceObject_t surface, dev_t *src, int pitch, int pixel_w, int pixel_h)
{
    unsigned int dim_x = blockDim.x * blockIdx.x + threadIdx.x;
    unsigned int dim_y = blockDim.y * blockIdx.y + threadIdx.y;

    if (dim_x < pixel_w && dim_y < pixel_h)
    {
        u_char r,g,b;
        r = *((u_char*)src + dim_y * pitch + dim_x * 4);
        g = *((u_char*)src + dim_y * pitch + dim_x * 4 + 1);
        b = *((u_char*)src + dim_y * pitch + dim_x * 4 + 2);
        uchar4 data = make_uchar4(r, g, b, 0xff);
        surf2Dwrite(data, surface, dim_x * sizeof(uchar4), dim_y);
    }
}

void RenderRGBAImageToSurface(ScGPUImage *image, hipSurfaceObject_t surface)
{
    dim3 block(8, 8);
    dim3 grid((image->m_pixel_width - 1) / block.x + 1, (image->m_pixel_height - 1) / block.y + 1);
    kernel_renderRGBA2Surface<<<grid, block>>>(surface, image->m_dev_addr,
                                                image->m_data_width, image->m_pixel_width, image->m_pixel_height);
}
